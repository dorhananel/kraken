
#include <hip/hip_runtime.h>
__global__ void vector_add(int *a, int *b, int length)
{
    int tx = threadIdx.x;

    if(tx < length) {
        a[tx] = a[tx] + b[tx];
    }
}

void kernel_wrapper(int *a, int *b, int length)
{

    int *a_d;
    int *b_d;
    int blockSize = 32;
    int numBlocks = (int)(length / blockSize);

    if(length % blockSize) {
        numBlocks++;
    }

    dim3 threads( blockSize, 1 );
    dim3 blocks( numBlocks, 1 );

    int byteLength = length * sizeof(int);

    hipMalloc( (void **)&a_d, byteLength );
    hipMalloc( (void **)&b_d, byteLength );

    hipMemcpy( a_d, a, byteLength, hipMemcpyHostToDevice );
    hipMemcpy( b_d, b, byteLength, hipMemcpyHostToDevice );

    vector_add<<< blocks, threads >>>( a, b , length);

    hipMemcpy( a, a_d, byteLength, hipMemcpyDeviceToHost );
    hipMemcpy( b, b_d, byteLength, hipMemcpyDeviceToHost );

    hipFree(a_d);
    hipFree(b_d);
}
